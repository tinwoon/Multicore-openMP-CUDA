#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <vector>
#include <utility>
#include <algorithm>
#include "DS_timer.h"

using namespace std;

#define PAGE (970)		//�Ѱ� 970
#define DAMPING (0.85)
#define BLOCK_SIZE (512)
#define NUM_BLOCKS (ceil((float)PAGE / BLOCK_SIZE))

float cal_PR(int, float *, int *, vector<int>);
void gen_graph(bool gr[][PAGE]);
void print_graph(bool gr[][PAGE]);
void printResult(float *, float *);

__global__ void analyize_graph(bool *_graph, int *_link_num, int *_link_index) {
	//������ ���̵�
	int tID = threadIdx.x + blockIdx.x * blockDim.x;

	//����ó��
	if (tID >= PAGE) return;

	//���������� ������ ��ũ �� �ľ�
	for (int i = 0; i < PAGE; i++) {
		int index = tID * PAGE + i;
		if (_graph[index])
			_link_num[tID]++;
	}

	//�������� ����Ű�� ��ũ �ε��� �ľ�
	int links = 0;
	for (int i = 0; i < PAGE; i++) {
		int index = i * PAGE + tID;
		if (_graph[index]) {
			_link_index[tID * PAGE + links] = i;
			links++;
		}
	}
}
__global__ void cal_PR(float *_PR, int *_link_num, int *_link_index, int *_cvg) {
	//������ ���̵�
	bool first = (threadIdx.x == 0) ? true : false;
	int tID = threadIdx.x + blockIdx.x * blockDim.x;

	__shared__ int cvg_part;
	if (first) cvg_part = 0;

	__syncthreads();

	float tmp = 0.0;
	//PR���
	if (tID < PAGE) {
		int present = tID * PAGE;
		int index = _link_index[present];

		while (index >= 0) {
			tmp += _PR[index] / _link_num[index];

			present++;
			index = _link_index[present];
		}
		tmp = ((1 - DAMPING) / (present % PAGE) + DAMPING * tmp);

		//����
		if (_PR[tID] == tmp)
			atomicAdd(&cvg_part, 1);
	}

	__syncthreads();
	_PR[tID] = tmp;

	if (first)
		atomicAdd(_cvg, cvg_part);
}
__global__ void reduct_PR(float *_PR, float *_sum) {
	//������ ���̵�
	int tID = threadIdx.x + blockIdx.x * blockDim.x;
	int ID = threadIdx.x;

	__shared__ float PR_tmp[BLOCK_SIZE];

	if (tID < PAGE) PR_tmp[ID] = _PR[tID];
	else PR_tmp[ID] = 0.0;
	__syncthreads();

	//�ջ�
	for (int span = BLOCK_SIZE / 2; span > 0; span /= 2) {
		if (ID < span) {
			PR_tmp[ID] += PR_tmp[ID + span];
		}
		__syncthreads();
	}

	if (ID == 0) {
		atomicAdd(_sum, PR_tmp[0]);
	}
}
__global__ void normal_PR(float *_PR, float *_sum) {
	//������ ���̵�
	int tID = threadIdx.x + blockIdx.x * blockDim.x;

	_PR[tID] /= *_sum;
}

int main() {
	//���� ����
	bool graph[PAGE][PAGE];
	int *link_num;
	vector<int> *link_index;
	float *PR, *PR_tmp, *PR_CUDA;

	//device����
	bool *d_graph;
	int *d_link_num, *d_link_index, *d_cvg;
	float *d_PR, *d_sum;
	
	//�Ҵ�
	link_num = new int[PAGE]; memset(link_num, 0, sizeof(int) * PAGE);
	link_index = new vector<int>[PAGE];
	PR = new float[PAGE];
	PR_tmp = new float[PAGE];
	PR_CUDA = new float[PAGE];

	//���� �ʱ�ȭ
	hipMalloc(&d_graph, sizeof(bool) * PAGE * PAGE);
	hipMalloc(&d_link_num, sizeof(int) * PAGE);
	hipMalloc(&d_link_index, sizeof(int) * PAGE * PAGE);
	hipMalloc(&d_PR, sizeof(float) * PAGE);
	hipMalloc(&d_cvg, sizeof(int) * 1);
	hipMalloc(&d_sum, sizeof(float) * 1);
	hipMemset(d_PR, 1.0, sizeof(float) * PAGE);
	hipMemset(d_link_num, 0, sizeof(int) * PAGE);
	hipMemset(d_link_index, -1, sizeof(int) * PAGE * PAGE);
	hipMemset(d_cvg, 0, sizeof(int) * 1);
	hipMemset(d_sum, 0.0, sizeof(float) * 1);

	//�ʱ� PR�� ��� 1.0
	for (int i = 0; i < PAGE; i++)
		PR[i] = 1.0;

	DS_timer timer(5);
	//Ÿ�̸� ����
	timer.setTimerName(0, (char*)"Serial");
	timer.setTimerName(1, (char*)"CUDA Total");
	timer.setTimerName(2, (char*)"Parallel Calculate");
	timer.setTimerName(3, (char*)"Data Transfer, H to D");
	timer.setTimerName(4, (char*)"Data Transfer, D to H");

	srand((unsigned int)time(0));

	//�׷��� ����
	gen_graph(graph);		
	
	/********** Serial **********/
	timer.onTimer(0);

	//�� ���������� ������ ��ũ �� �ľ�
	for (int i = 0; i < PAGE; i++)
		for (int j = 0; j < PAGE; j++)
			if (graph[i][j])
				link_num[i]++;

	//�� �������� ����Ű�� ��ũ �ε��� �ľ�
	for (int i = 0; i < PAGE; i++)
		for (int j = 0; j < PAGE; j++)
			if (graph[j][i])
				link_index[i].push_back(j);

	//PR���
	int cvg = 0;
	float sum = 0.0;

	while (cvg != PAGE) {
		cvg = 0;
		sum = 0.0;

		//������ PR ���
		for (int i = 0; i < PAGE; i++) {
			PR_tmp[i] = cal_PR(i, PR, link_num, link_index[i]);
			sum += PR_tmp[i];
		}

		//PR ����
		for (int i = 0; i < PAGE; i++) {
			if (PR[i] == PR_tmp[i]) cvg++;
			else PR[i] = PR_tmp[i];
		}
	}

	//����ȭ
	for (int i = 0; i < PAGE; i++) {
		PR[i] /= sum;
	}

	timer.offTimer(0);

	/********** CUDA **********/
	timer.onTimer(1);

	//Data Transfer HostToDevice
	timer.onTimer(3);
	hipMemcpy(d_graph, graph, sizeof(bool) * PAGE * PAGE, hipMemcpyHostToDevice);
	timer.offTimer(3);

	dim3 blockDim(BLOCK_SIZE);
	dim3 gridDim(NUM_BLOCKS);

	//Kernel Call
	timer.onTimer(2);
	analyize_graph << <gridDim, blockDim >> > (d_graph, d_link_num, d_link_index);
	hipDeviceSynchronize();
	timer.offTimer(2);

	cvg = 0;
	while (cvg != PAGE) {
		timer.onTimer(2);
		hipMemset(d_cvg, 0, sizeof(int) * 1);

		cal_PR << <gridDim, blockDim>> > (d_PR, d_link_num, d_link_index, d_cvg);
		hipDeviceSynchronize();
		timer.offTimer(2);

		timer.onTimer(4);
		hipMemcpy(&cvg, d_cvg, sizeof(int) * 1, hipMemcpyDeviceToHost);
		timer.offTimer(4);
	}

	timer.onTimer(2);
	reduct_PR << <gridDim, blockDim >> > (d_PR, d_sum);
	normal_PR << <gridDim, blockDim >> > (d_PR, d_sum);
	hipDeviceSynchronize();
	timer.offTimer(2);

	//Data Transfer DeviceToHost
	timer.onTimer(4);
	hipMemcpy(PR_CUDA, d_PR, sizeof(float) * PAGE, hipMemcpyDeviceToHost);
	timer.offTimer(4);

	timer.offTimer(1);

	printResult(PR, PR_CUDA);
	timer.printTimer();
}

float cal_PR(int page, float *pageRank, int *link_num, vector<int> link_index) {
	float tmp = 0.0;
	int size = link_index.size();

	for (int i = 0; i < size; i++) {
		int index = link_index[i];

		tmp += pageRank[index] / link_num[index];
	}

	return ((1 - DAMPING) / size + DAMPING * tmp);
}
void gen_graph(bool gr[][PAGE]) {
	for (int i = 0; i < PAGE; i++)
		for (int j = 0; j < PAGE; j++) {
			if (i == j) gr[i][j] = 0;
			else gr[i][j] = (rand() % 2);
		}
}
void print_graph(bool gr[][PAGE]){
	for (int i = 0; i < PAGE; i++) {
		for (int j = 0; j < PAGE; j++)
			printf("%d ", gr[i][j]);
		printf("\n");
	}
}
void printResult(float *A, float *B) {
	vector<pair<float, int>> PR_A;
	vector<pair<float, int>> PR_B;

	for (int i = 0; i < PAGE; i++) {
		PR_A.push_back(pair<float, int>(A[i], i));
		PR_B.push_back(pair<float, int>(B[i], i));
	}

	sort(PR_A.begin(), PR_A.end());
	sort(PR_B.begin(), PR_B.end());

	int count = 25;

	printf("���� %dpage PageRank ��\n", count);
	printf("=======================================================================\n");
	for (int i = 0; i < count; i++) {
		pair<float, int> a = PR_A.back();
		pair<float, int> b = PR_B.back();
		PR_A.pop_back();
		PR_B.pop_back();

		printf("%2d���� : Serial - [%3d] %f, CUDA - [%3d] %f\n", 
			i + 1, a.second, a.first, b.second, b.first);
	}
	printf("...\n");
	printf("=======================================================================\n");
}